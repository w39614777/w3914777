#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
int main(void){
    purelowprecision *eta1,*eta2,*eta1_lap,*eta2_lap,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(half)*dimX*dimY));
    #ifdef HALF
        dim3 blocks(unitx,unity);
        dim3 grids(1,1,unitdimX*unitdimY);
        dataprepare_half<<<grids,blocks>>>(eta1,eta2);
        hipDeviceSynchronize();
        #define TOLOW __float2half
        #define FILENAME "halfresult.csv"
    #else
        dim3 blocks(uxd2,unity);
        dim3 grids(1,1,unitdimX*unitdimY);
        dataprepare_half2<<<grids,blocks>>>(eta1,eta2);
        hipDeviceSynchronize();
        #define TOLOW __float2half2_rn
        #define FILENAME "half2result.csv"
    #endif
    purelowprecision hmobil=TOLOW(mobil);
    purelowprecision hgrcoef=TOLOW(grcoef);
    purelowprecision hdx=TOLOW(dx);
    purelowprecision hdy=TOLOW(dy);
    purelowprecision hdxdy=TOLOW(dxdy);
    purelowprecision height=TOLOW(8.0);
    purelowprecision hdtime=TOLOW(dtime);
    purelowprecision hone=TOLOW(1.0);
    purelowprecision htwo=TOLOW(2.0);
    for(int i=0;i<timesteps;i++){
        kernel1_lowpure<<<grids,blocks>>>(eta1,eta2,eta1_lap,dfdeta1,eta1_out,hone,htwo,height,hdtime,hmobil,hgrcoef,hdxdy);
        kernel1_lowpure<<<grids,blocks>>>(eta2,eta1,eta2_lap,dfdeta2,eta2_out,hone,htwo,height,hdtime,hmobil,hgrcoef,hdxdy);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
    }
    float *feta2;
    CHECK_ERROR(hipMallocManaged((void**)&feta2,sizeof(float)*dimX*dimY));
    purelow2high_aftercomputing<<<grids,blocks>>>(eta2,feta2);
    hipDeviceSynchronize();
    writetocsv(FILENAME,feta2,dimX,dimY);
    cout<<FILENAME<<endl;
    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    CHECK_ERROR(hipFree(feta2));
}