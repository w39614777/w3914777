#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
int main(int argc,char* argv[]){
    highprecision *eta1,*eta2,*eta1_lap,*eta2_lap,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));   
    lowprecision *heta1,*heta2,*heta1_lap,*heta2_lap,*hdfdeta1,*hdfdeta2,*heta1_out,*heta2_out;
    int *type_curr;
    CHECK_ERROR(hipMallocManaged((void**)&heta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_curr,sizeof(int)*unitdimX*unitdimY));
    highprecision *eta2_last;
    CHECK_ERROR(hipMallocManaged((void**)&eta2_last,sizeof(highprecision)*dimX*dimY));
    #ifdef AMSTENCIL
        int *type_old;
        highprecision *max_diff;
        CHECK_ERROR(hipMallocManaged((void**)&type_old,sizeof(int)*unitdimX*unitdimY));
        CHECK_ERROR(hipMallocManaged((void**)&max_diff,sizeof(highprecision)*unitdimX*unitdimY));
    #endif
    dim3 blocks_prepare_high(unitx,unity);
    dim3 grids_prepare_high(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids_prepare_high,blocks_prepare_high>>>(eta1,eta2);
    hipDeviceSynchronize();
    #ifdef AMSTENCIL
        dim3 blocks_prepare_half2(uxd2,unity);
        dim3 grids_prepare_half2(1,1,unitdimX*unitdimY);
        dataprepare_half2<<<grids_prepare_half2,blocks_prepare_half2>>>(heta1,heta2);
        hipDeviceSynchronize();
        #define TOLOW __float2half2_rn
        #ifdef Monitor2
            
            for(int i=0;i<dimX*dimY;i++){
                eta2_last[i]=eta2[i]==0?1.0:0.0;
            }
        #endif
        dim3 blocks_detect(unitdimX>32?32:unitdimX,unitdimY>32?32:unitdimY);
        dim3 grids_detect(unitdimX>32?unitdimX/32:1,unitdimY>32?unitdimY/32:1);
        dim3 datasychronduring_blocks(uxd2,unity);
        dim3 datasychronduring_grids(1,1,unitdimX*unitdimY);
    #else
        dim3 blocks_prepare_half(unitx,unity);
        dim3 grids_prepare_half(1,1,unitdimX*unitdimY);
        dataprepare_half<<<grids_prepare_half,blocks_prepare_half>>>(heta1,heta2);
        hipDeviceSynchronize();
        #define TOLOW __float2half
    #endif
    lowprecision hmobil=TOLOW(mobil);
    lowprecision hgrcoef=TOLOW(grcoef);
    lowprecision hdx=TOLOW(dx);
    lowprecision hdy=TOLOW(dy);
    lowprecision hdxdy=TOLOW(dxdy);
    lowprecision height=TOLOW(8.0);
    lowprecision hdtime=TOLOW(dtime);
    lowprecision hone=TOLOW(1.0);
    lowprecision htwo=TOLOW(2.0);
    dim3 blocks_mix(uxd2,unity);
    dim3 grids_mix(2,1,unitdimX*unitdimY);
    hipEvent_t startmix,stopmix;float elapsedmix;
    for(int i=0;i<timesteps;i++){
        if(i==5){
            CHECK_ERROR(hipEventCreate(&startmix));
            CHECK_ERROR(hipEventCreate(&stopmix));
            CHECK_ERROR(hipEventRecord(startmix,0));
        }
        #ifdef AMSTENCIL
            if(i%10==0){
                #ifdef Monitor1
                get_max_diff1<<<grids_detect,blocks_detect>>>(eta2,max_diff);
                #endif
                #ifdef Monitor2
                get_max_diff2<<<grids_detect,blocks_detect>>>(eta2_last,eta2,max_diff);
                #endif
                get_type<<<grids_detect,blocks_detect>>>(max_diff,type_old,type_curr);
                hipDeviceSynchronize();
                // writetocsv("max"+to_string(i)+".csv",max_diff,unitdimX,unitdimY);
                // writetocsv("type"+to_string(i)+".csv",type_curr,unitdimX,unitdimY);
                data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(eta1,eta2,heta1,heta2,type_old,type_curr);
                hipDeviceSynchronize();
            }
        #endif
        kernel1_mix<<<grids_mix,blocks_mix>>>(eta1,eta2_last,eta2,eta1_lap,dfdeta1,eta1_out,heta1,heta2,heta1_lap,hdfdeta1,heta1_out,height,hdxdy,hone,htwo,hdtime,hmobil,hgrcoef,type_curr,i,1);
        kernel1_mix<<<grids_mix,blocks_mix>>>(eta2,eta2_last,eta1,eta2_lap,dfdeta2,eta2_out,heta2,heta1,heta2_lap,hdfdeta2,heta2_out,height,hdxdy,hone,htwo,hdtime,hmobil,hgrcoef,type_curr,i,2);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
    }
    if(timesteps>5){
        CHECK_ERROR(hipEventRecord(stopmix,0));
        CHECK_ERROR(hipEventSynchronize(stopmix));
        CHECK_ERROR(hipEventElapsedTime(&elapsedmix,startmix,stopmix));
        CHECK_ERROR(hipEventDestroy(startmix));
        CHECK_ERROR(hipEventDestroy(stopmix));
    }
    ofstream ftime("time_tmp.csv");
    ftime<<elapsedmix;
    ftime.close();
    //每个程序跑五次测试性能，最后一次记录计算结果
    if(string(argv[1])=="4"){
        string path=string(argv[2]);
        data_sychro_aftercomputation<<<grids_mix,blocks_mix>>>(eta2,heta2,type_curr);
        hipDeviceSynchronize();
        writetocsv(path,eta2,dimX,dimY);
    }
    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    CHECK_ERROR(hipFree(heta1));
    CHECK_ERROR(hipFree(heta2));
    CHECK_ERROR(hipFree(heta1_out));
    CHECK_ERROR(hipFree(heta2_out));
    CHECK_ERROR(hipFree(heta1_lap));
    CHECK_ERROR(hipFree(heta2_lap));
    CHECK_ERROR(hipFree(hdfdeta1));
    CHECK_ERROR(hipFree(hdfdeta2));
    CHECK_ERROR(hipFree(type_curr));
    CHECK_ERROR(hipFree(eta2_last));
    #ifdef AMSTENCIL
        CHECK_ERROR(hipFree(type_old));
        CHECK_ERROR(hipFree(max_diff));
    #endif
}