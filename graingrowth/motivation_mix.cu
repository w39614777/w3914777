#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
#include <queue>
int main(int argc ,char* argv[]){
    highprecision *eta1,*eta2,*eta1_lap,*eta2_lap,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));   
    lowprecision *heta1,*heta2,*heta1_lap,*heta2_lap,*hdfdeta1,*hdfdeta2,*heta1_out,*heta2_out;
    int *type_curr;
    CHECK_ERROR(hipMallocManaged((void**)&heta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_curr,sizeof(int)*unitdimX*unitdimY));
    highprecision *eta2_last;
    CHECK_ERROR(hipMallocManaged((void**)&eta2_last,sizeof(highprecision)*dimX*dimY));
    #ifdef AMSTENCIL
        highprecision *max_diff;
        CHECK_ERROR(hipMallocManaged((void**)&max_diff,sizeof(highprecision)*unitdimX*unitdimY));
        #define TOLOW __float2half2_rn
        dim3 blocks_detect(unitdimX>32?32:unitdimX,unitdimY>32?32:unitdimY);
        dim3 grids_detect(unitdimX>32?unitdimX/32:1,unitdimY>32?unitdimY/32:1);
    #else
        #define TOLOW __float2half
    #endif
    lowprecision hmobil=TOLOW(mobil);
    lowprecision hgrcoef=TOLOW(grcoef);
    lowprecision hdx=TOLOW(dx);
    lowprecision hdy=TOLOW(dy);
    lowprecision hdxdy=TOLOW(dxdy);
    lowprecision height=TOLOW(8.0);
    lowprecision hdtime=TOLOW(dtime);
    lowprecision hone=TOLOW(1.0);
    lowprecision htwo=TOLOW(2.0);
    dim3 blocks_prepare_high(unitx,unity);
    dim3 grids_prepare_high(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids_prepare_high,blocks_prepare_high>>>(eta1,eta2);
    hipDeviceSynchronize();

    dim3 blocks_pure(unitx,unity);
    dim3 grids_pure(1,1,unitdimX*unitdimY);
    for(int i=0;i<timesteps;i++){
        kernel1_pure<<<grids_pure,blocks_pure>>>(eta1,eta2,eta1_lap,dfdeta1,eta1_out);
        kernel1_pure<<<grids_pure,blocks_pure>>>(eta2,eta1,eta2_lap,dfdeta2,eta2_out);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        #if ((defined Monitor2)&&(defined AMSTENCIL))
            if(i==timesteps-10){
                motivation_monitor2_datasychr<<<grids_pure,blocks_pure>>>(eta2,eta2_last);
                hipDeviceSynchronize();
            }
        #endif        
    }
    dim3 blocks_mix(uxd2,unity);
    dim3 grids_mix(2,1,unitdimX*unitdimY);
    highdata_to_low<<<grids_mix,blocks_mix>>>(eta1,heta1);
    highdata_to_low<<<grids_mix,blocks_mix>>>(eta2,heta2);
    #ifdef AMSTENCIL
        int highprecisionnum=unitdimX*unitdimY-unitdimX*unitdimY*ratio,sum=0;
        #ifdef Monitor1
            get_max_diff1<<<grids_detect,blocks_detect>>>(eta2,max_diff);
        #endif
        #ifdef Monitor2
            get_max_diff2<<<grids_detect,blocks_detect>>>(eta2_last,eta2,max_diff);
        #endif
        hipDeviceSynchronize();
        
        int *index;
        index=new int[unitdimX*unitdimY];
        BubbleSort(max_diff,unitdimX*unitdimY,index);
        for(int j=0;j<unitdimX*unitdimY;j++)type_curr[j]=1;
        queue<int> queue_index;
        for(int j=0;max_diff[j]!=0;j++){
            type_curr[index[j]]=2;
            sum++;
            queue_index.push(index[j]);
            if(sum>=highprecisionnum)break;
        }
        while(!queue_index.empty()){
            int center_index=queue_index.front();
            queue_index.pop();
            for(int direct=1;direct<=8;direct++){
                if(type_curr[get_neibour(center_index,direct,1)]!=2){
                    sum++;
                    type_curr[get_neibour(center_index,direct,1)]=2;
                    queue_index.push(get_neibour(center_index,direct,1));
                    if(sum>=highprecisionnum)break;
                }
            }
            if(sum>=highprecisionnum)break;
        }
        delete []index;
    #endif
    for(int i=timesteps;i<timesteps+50;i++){
        kernel1_mix<<<grids_mix,blocks_mix>>>(eta1,eta2_last,eta2,eta1_lap,dfdeta1,eta1_out,heta1,heta2,heta1_lap,hdfdeta1,heta1_out,height,hdxdy,hone,htwo,hdtime,hmobil,hgrcoef,type_curr,i,1);
        kernel1_mix<<<grids_mix,blocks_mix>>>(eta2,eta2_last,eta1,eta2_lap,dfdeta2,eta2_out,heta2,heta1,heta2_lap,hdfdeta2,heta2_out,height,hdxdy,hone,htwo,hdtime,hmobil,hgrcoef,type_curr,i,2);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);

    }
    data_sychro_aftercomputation<<<grids_mix,blocks_mix>>>(eta2,heta2,type_curr);
    hipDeviceSynchronize();
    string paths=string(argv[1]);
    writetocsv(paths,eta2,dimX,dimY);

    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    CHECK_ERROR(hipFree(heta1));
    CHECK_ERROR(hipFree(heta2));
    CHECK_ERROR(hipFree(heta1_out));
    CHECK_ERROR(hipFree(heta2_out));
    CHECK_ERROR(hipFree(heta1_lap));
    CHECK_ERROR(hipFree(heta2_lap));
    CHECK_ERROR(hipFree(hdfdeta1));
    CHECK_ERROR(hipFree(hdfdeta2));
    CHECK_ERROR(hipFree(type_curr));
    CHECK_ERROR(hipFree(eta2_last));
    #ifdef AMSTENCIL
        CHECK_ERROR(hipFree(max_diff));
    #endif
    return 0;
}