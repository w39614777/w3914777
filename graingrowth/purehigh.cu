#include "function.h"
#include "tools.h"
int main(int argc,char* argv[]){
    highprecision *eta1,*eta2,*eta1_lap,*eta2_lap,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));
    dim3 blocks(unitx,unity);
    dim3 grids(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids,blocks>>>(eta1,eta2);
    hipDeviceSynchronize();
    #ifdef Motivation
         timesteps=timesteps+50;
    #endif
    #ifdef End2end
        hipEvent_t start,stop;float elapsed;
    #endif
    for(int i=0;i<timesteps;i++){
        #ifdef End2end
            if(i==5){
                CHECK_ERROR(hipEventCreate(&start));
                CHECK_ERROR(hipEventCreate(&stop));
                CHECK_ERROR(hipEventRecord(start,0));
            }
        #endif
        kernel1_pure<<<grids,blocks>>>(eta1,eta2,eta1_lap,dfdeta1,eta1_out);
        kernel1_pure<<<grids,blocks>>>(eta2,eta1,eta2_lap,dfdeta2,eta2_out);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
    }
    #ifdef End2end
        if(timesteps>5){
            CHECK_ERROR(hipEventRecord(stop,0));
            CHECK_ERROR(hipEventSynchronize(stop));
            CHECK_ERROR(hipEventElapsedTime(&elapsed,start,stop));
            CHECK_ERROR(hipEventDestroy(start));
            CHECK_ERROR(hipEventDestroy(stop));
        }
        ofstream ftime("time_tmp.csv");
        ftime<<elapsed;
        ftime.close();
    #endif
    #ifdef End2end
        if(string(argv[1])=="4"){
            string path=string(argv[2]);
            writetocsv(path,eta2,dimX,dimY);
        }
    #endif
    #ifdef Motivation
        string path=string(argv[1]);
        writetocsv(path,eta2,dimX,dimY);
    #endif
    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    return 0;
}