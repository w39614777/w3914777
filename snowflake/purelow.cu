#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
int main(void){
    purelowprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(half)*dimX*dimY));
    #ifdef HALF
        dim3 blocks(unitx,unity);
        dim3 grids(1,1,unitdimX*unitdimY);
        dataprepare_half<<<grids,blocks>>>(phi);
        hipDeviceSynchronize();
        #define TOLOW __float2half
        #define FILENAME "halfresult.csv"
    #else
        dim3 blocks(uxd2,unity);
        dim3 grids(1,1,unitdimX*unitdimY);
        dataprepare_half2<<<grids,blocks>>>(phi);
        hipDeviceSynchronize();
        #define TOLOW __float2half2_rn
        #define FILENAME "half2result.csv"
    #endif
    purelowprecision hdxdy=TOLOW(dxdy);
    purelowprecision hdym2=TOLOW(dy*2);
    purelowprecision hdxm2=TOLOW(dx*2);
    purelowprecision height=TOLOW(8.0);
    purelowprecision htheta0=TOLOW(theta0);
    purelowprecision haniso=TOLOW(aniso);
    purelowprecision hdelta=TOLOW(delta);
    purelowprecision hone=TOLOW(1.0);
    purelowprecision hepsilonb=TOLOW(epsilonb);
    purelowprecision halpha=TOLOW(alpha);
    purelowprecision hpi=TOLOW(pi);
    purelowprecision hgama=TOLOW(gama);
    purelowprecision hkappa=TOLOW(kappa);
    purelowprecision hzpf=TOLOW(0.5);
    purelowprecision hteq=TOLOW(teq);
    purelowprecision hdtime=TOLOW(dtime);
    purelowprecision htau=TOLOW(tau);
    hipEvent_t start,stop;float elapsed;
    for(int i=0;i<timesteps;i++){
        if(i==5){
            CHECK_ERROR(hipEventCreate(&start));
            CHECK_ERROR(hipEventCreate(&stop));
            CHECK_ERROR(hipEventRecord(start,0));
        }
        kernel1_purelow<<<grids,blocks>>>(phi,phi_lap,tempr,tempr_lap,phidx,phidy,epsilon,epsilon_deri,hdxdy,htheta0,haniso,hone,hdxm2,hdym2,hdelta,hepsilonb,height);
        kernel2_purelow<<<grids,blocks>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hdym2,hdxm2,hgama,hteq,halpha,hpi,hone,hzpf,hdtime,htau,hkappa);
        hipDeviceSynchronize();
    }
    if(timesteps>5){
        CHECK_ERROR(hipEventRecord(stop,0));
        CHECK_ERROR(hipEventSynchronize(stop));
        CHECK_ERROR(hipEventElapsedTime(&elapsed,start,stop));
        CHECK_ERROR(hipEventDestroy(start));
        CHECK_ERROR(hipEventDestroy(stop));
    }
    ofstream ftime("time_tmp.csv");
    ftime<<elapsed;
    ftime.close();
    float *fphi;
    CHECK_ERROR(hipMallocManaged((void**)&fphi,sizeof(float)*dimX*dimY));
    purelow2high_aftercomputing<<<grids,blocks>>>(phi,fphi);
    hipDeviceSynchronize();
    writetocsv(FILENAME,fphi,dimX,dimY);
    cout<<FILENAME<<endl;
    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    CHECK_ERROR(hipFree(fphi));
    return 0;
}