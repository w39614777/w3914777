#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
int main(int argc,char* argv[]){
    highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
    lowprecision *hphi,*hphi_lap,*htempr,*htempr_lap,*hphidx,*hphidy,*hepsilon,*hepsilon_deri;
    int *type_new;
    CHECK_ERROR(hipMallocManaged((void**)&hphi,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphi_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidx,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon_deri,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_new,sizeof(int)*unitdimX*unitdimY));
    highprecision *phi_old;
    CHECK_ERROR(hipMallocManaged((void**)&phi_old,sizeof(highprecision)*dimX*dimY));
    int *type_old;
    highprecision *max_diff;
    CHECK_ERROR(hipMallocManaged((void**)&type_old,sizeof(int)*unitdimX*unitdimY));
    CHECK_ERROR(hipMallocManaged((void**)&max_diff,sizeof(highprecision)*unitdimX*unitdimY));
    dim3 blocks_prepare_high(unitx,unity);
    dim3 grids_prepare_high(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids_prepare_high,blocks_prepare_high>>>(phi);
    hipDeviceSynchronize();
    dim3 blocks_prepare_half2(uxd2,unity);
    dim3 grids_prepare_half2(1,1,unitdimX*unitdimY);
    dataprepare_half2<<<grids_prepare_half2,blocks_prepare_half2>>>(hphi);
    hipDeviceSynchronize();
    #define TOLOW __float2half2_rn
    #ifdef Monitor2
        for(int i=0;i<dimX*dimY;i++){
            phi_old[i]=phi[i]==0?1.0:0.0;
        }
    #endif
    dim3 blocks_detect(unitdimX>32?32:unitdimX,unitdimY>32?32:unitdimY);
    dim3 grids_detect(unitdimX>32?unitdimX/32:1,unitdimY>32?unitdimY/32:1);
    dim3 datasychronduring_blocks(uxd2,unity);
    dim3 datasychronduring_grids(1,1,unitdimX*unitdimY);
    lowprecision hdxdy=TOLOW(dxdy);
    lowprecision hdym2=TOLOW(dy*2);
    lowprecision hdxm2=TOLOW(dx*2);
    lowprecision height=TOLOW(8.0);
    lowprecision htheta0=TOLOW(theta0);
    lowprecision haniso=TOLOW(aniso);
    lowprecision hdelta=TOLOW(delta);
    lowprecision hone=TOLOW(1.0);
    lowprecision hepsilonb=TOLOW(epsilonb);
    lowprecision halpha=TOLOW(alpha);
    lowprecision hpi=TOLOW(pi);
    lowprecision hgama=TOLOW(gama);
    lowprecision hkappa=TOLOW(kappa);
    lowprecision hzpf=TOLOW(0.5);
    lowprecision hteq=TOLOW(teq);
    lowprecision hdtime=TOLOW(dtime);
    lowprecision htau=TOLOW(tau);
    dim3 blocks_mix(uxd2,unity);
    dim3 grids_mix(2,1,unitdimX*unitdimY);
    #ifdef Monitor1
        get_max_diff1<<<grids_detect,blocks_detect>>>(phi,max_diff);
    #endif
    #ifdef Monitor2
        get_max_diff2<<<grids_detect,blocks_detect>>>(phi_old,phi,max_diff);
        monitor2_lastdata_store<<<grids_mix,blocks_mix>>>(phi,phi_old);
    #endif
    get_type<<<grids_detect,blocks_detect>>>(max_diff,type_old,type_new);
    hipDeviceSynchronize();
    data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(phi,tempr,hphi,htempr,type_old,type_new);
    hipDeviceSynchronize();
    for(int i=0;i<5;i++){
        kernel1_mix<<<grids_mix,blocks_mix>>>(phi,phi_old,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,htempr,htempr_lap,hphidx,hphidy,hepsilon,hepsilon_deri,hdxdy,htheta0,haniso,hone,hdxm2,hdym2,hdelta,hepsilonb,height,type_new,i);
        kernel1_conversion<<<grids_mix,blocks_mix>>>(epsilon,epsilon_deri,phidx,phidy,hphidx,hphidy,hepsilon,hepsilon_deri,type_new,i);
        mix_kernel2<<<grids_mix,blocks_mix>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,hepsilon,hepsilon_deri,hphidx,hphidy,htempr,htempr_lap,hdym2,hdxm2,hgama,hteq,halpha,hpi,hone,hzpf,hdtime,htau,hkappa,type_new,i);
        kernel2_conversion<<<grids_mix,blocks_mix>>>(phi,tempr,hphi,htempr,type_new,i);
        hipDeviceSynchronize();                 
    }
    hipEvent_t kernel_start,kernel_end;float elapsed_kernel=0;
    CHECK_ERROR(hipEventCreate(&kernel_start));
    CHECK_ERROR(hipEventCreate(&kernel_end));
    float total_time=0;
    for(int i=5;i<timesteps;i++){
        if(i%10==0){
            #ifdef Monitor1
                get_max_diff1<<<grids_detect,blocks_detect>>>(phi,max_diff);
            #endif
            #ifdef Monitor2
                get_max_diff2<<<grids_detect,blocks_detect>>>(phi_old,phi,max_diff);
                hipDeviceSynchronize();
                monitor2_lastdata_store<<<grids_mix,blocks_mix>>>(phi,phi_old);
            #endif
            hipDeviceSynchronize();
            get_type<<<grids_detect,blocks_detect>>>(max_diff,type_old,type_new);
            hipDeviceSynchronize();
            data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(phi,tempr,hphi,htempr,type_old,type_new);
            hipDeviceSynchronize();
        }
        CHECK_ERROR(hipEventRecord(kernel_start,0));
        kernel1_mix<<<grids_mix,blocks_mix>>>(phi,phi_old,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,htempr,htempr_lap,hphidx,hphidy,hepsilon,hepsilon_deri,hdxdy,htheta0,haniso,hone,hdxm2,hdym2,hdelta,hepsilonb,height,type_new,i);
        hipDeviceSynchronize();
        CHECK_ERROR(hipEventRecord(kernel_end,0));
        CHECK_ERROR(hipEventSynchronize(kernel_end));
        CHECK_ERROR(hipEventElapsedTime(&elapsed_kernel,kernel_start,kernel_end));
        total_time+=elapsed_kernel;

        kernel1_conversion<<<grids_mix,blocks_mix>>>(epsilon,epsilon_deri,phidx,phidy,hphidx,hphidy,hepsilon,hepsilon_deri,type_new,i);
        hipDeviceSynchronize();


        CHECK_ERROR(hipEventRecord(kernel_start,0));
        mix_kernel2<<<grids_mix,blocks_mix>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,hepsilon,hepsilon_deri,hphidx,hphidy,htempr,htempr_lap,hdym2,hdxm2,hgama,hteq,halpha,hpi,hone,hzpf,hdtime,htau,hkappa,type_new,i);
        hipDeviceSynchronize();
        CHECK_ERROR(hipEventRecord(kernel_end,0));
        CHECK_ERROR(hipEventSynchronize(kernel_end));
        CHECK_ERROR(hipEventElapsedTime(&elapsed_kernel,kernel_start,kernel_end));
        total_time+=elapsed_kernel;
        
        kernel2_conversion<<<grids_mix,blocks_mix>>>(phi,tempr,hphi,htempr,type_new,i);
        hipDeviceSynchronize();  
    }
    CHECK_ERROR(hipEventDestroy(kernel_start));
    CHECK_ERROR(hipEventDestroy(kernel_end));
    ofstream ftime("time_tmp.csv");
    ftime<<total_time;
    ftime.close();
    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    CHECK_ERROR(hipFree(hphi));
    CHECK_ERROR(hipFree(hphi_lap));
    CHECK_ERROR(hipFree(htempr));
    CHECK_ERROR(hipFree(htempr_lap));
    CHECK_ERROR(hipFree(hphidx));
    CHECK_ERROR(hipFree(hphidy));
    CHECK_ERROR(hipFree(hepsilon));
    CHECK_ERROR(hipFree(hepsilon_deri));
    CHECK_ERROR(hipFree(type_new));
    CHECK_ERROR(hipFree(phi_old));
    CHECK_ERROR(hipFree(type_old));
    CHECK_ERROR(hipFree(max_diff));
    return 0;
}