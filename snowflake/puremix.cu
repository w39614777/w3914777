#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
int main(void){
    highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
    lowprecision *hphi,*hphi_lap,*htempr,*htempr_lap,*hphidx,*hphidy,*hepsilon,*hepsilon_deri;
    int *type_new;
    CHECK_ERROR(hipMallocManaged((void**)&hphi,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphi_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidx,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon_deri,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_new,sizeof(int)*unitdimX*unitdimY));
    highprecision *phi_old;
    CHECK_ERROR(hipMallocManaged((void**)&phi_old,sizeof(highprecision)*dimX*dimY));
    #ifdef AMSTENCIL
        for(int i=0;i<unitdimX*unitdimY;i++){
            // type_new[i]=2;
            if(i<unitNums*ratio)type_new[i]=1;
            else type_new[i]=2;
        }
    #endif  
        dim3 blocks_preparehigh(unitx,unity);
        dim3 grids_preparehigh(1,1,unitdimX*unitdimY);
        dataprepare_high<<<grids_preparehigh,blocks_preparehigh>>>(phi);  

     #ifdef AMSTENCIL
        dim3 blocks_prepare_half2(uxd2,unity);
        dim3 grids_prepare_half2(1,1,unitdimX*unitdimY);
        dataprepare_half2<<<grids_prepare_half2,blocks_prepare_half2>>>(hphi);
        hipDeviceSynchronize();
        #define TOLOW __float2half2_rn

    #else
        dim3 blocks_prepare_half(unitx,unity);
        dim3 grids_prepare_half(1,1,unitdimX*unitdimY);
        dataprepare_half<<<grids_prepare_half,blocks_prepare_half>>>(hphi);
        hipDeviceSynchronize();
        #define TOLOW __float2half
    #endif
    lowprecision hdxdy=TOLOW(dxdy);
    lowprecision hdym2=TOLOW(dy*2);
    lowprecision hdxm2=TOLOW(dx*2);
    lowprecision height=TOLOW(8.0);
    lowprecision htheta0=TOLOW(theta0);
    lowprecision haniso=TOLOW(aniso);
    lowprecision hdelta=TOLOW(delta);
    lowprecision hone=TOLOW(1.0);
    lowprecision hepsilonb=TOLOW(epsilonb);
    lowprecision halpha=TOLOW(alpha);
    lowprecision hpi=TOLOW(pi);
    lowprecision hgama=TOLOW(gama);
    lowprecision hkappa=TOLOW(kappa);
    lowprecision hzpf=TOLOW(0.5);
    lowprecision hteq=TOLOW(teq);
    lowprecision hdtime=TOLOW(dtime);
    lowprecision htau=TOLOW(tau);  
    dim3 blocks_mix(uxd2,unity);
    dim3 grids_mix(2,1,unitdimX*unitdimY);
    for(int i=0;i<timesteps;i++){
        kernel1_mix<<<grids_mix,blocks_mix>>>(phi,phi_old,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,htempr,htempr_lap,hphidx,hphidy,hepsilon,hepsilon_deri,hdxdy,htheta0,haniso,hone,hdxm2,hdym2,hdelta,hepsilonb,height,type_new,i);
        mix_kernel2<<<grids_mix,blocks_mix>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,hepsilon,hepsilon_deri,hphidx,hphidy,htempr,htempr_lap,hdym2,hdxm2,hgama,hteq,halpha,hpi,hone,hzpf,hdtime,htau,hkappa,type_new,i);
        hipDeviceSynchronize();
    }
    // #ifdef AMSTENCIL
    //     dim3 blocks(uxd2,unity);
    //     dim3 grids(1,1,unitdimX*unitdimY);
    // #else
    //     dim3 blocks(unitx,unity);
    //     dim3 grids(1,1,unitdimX*unitdimY);
    // #endif
    // purelow2high_aftercomputing<<<grids,blocks>>>(hphi,phi);
    // hipDeviceSynchronize();
    data_sychro_aftercomputation<<<grids_mix,blocks_mix>>>(phi,hphi,type_new);
    hipDeviceSynchronize();
    #ifdef AMSTENCIL
        writetocsv("amstencil.csv",phi,dimX,dimY);
    #endif
    #ifdef GRAM1
        writetocsv("gram1.csv",phi,dimX,dimY);
    #endif
    #ifdef GRAM2
        writetocsv("gram2.csv",phi,dimX,dimY);
    #endif
    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    CHECK_ERROR(hipFree(hphi));
    CHECK_ERROR(hipFree(hphi_lap));
    CHECK_ERROR(hipFree(htempr));
    CHECK_ERROR(hipFree(htempr_lap));
    CHECK_ERROR(hipFree(hphidx));
    CHECK_ERROR(hipFree(hphidy));
    CHECK_ERROR(hipFree(hepsilon));
    CHECK_ERROR(hipFree(hepsilon_deri));
    CHECK_ERROR(hipFree(type_new));
    CHECK_ERROR(hipFree(phi_old));
    return 0;
}
