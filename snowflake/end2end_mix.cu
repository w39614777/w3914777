#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
int main(int argc,char* argv[]){
    highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
    lowprecision *hphi,*hphi_lap,*htempr,*htempr_lap,*hphidx,*hphidy,*hepsilon,*hepsilon_deri;
    int *type_new;
    CHECK_ERROR(hipMallocManaged((void**)&hphi,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphi_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidx,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon_deri,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_new,sizeof(int)*unitdimX*unitdimY));
    highprecision *phi_old;
    CHECK_ERROR(hipMallocManaged((void**)&phi_old,sizeof(highprecision)*dimX*dimY));
    #ifdef AMSTENCIL
        int *type_old;
        highprecision *max_diff;
        CHECK_ERROR(hipMallocManaged((void**)&type_old,sizeof(int)*unitdimX*unitdimY));
        CHECK_ERROR(hipMallocManaged((void**)&max_diff,sizeof(highprecision)*unitdimX*unitdimY));
    #endif
    dim3 blocks_prepare_high(unitx,unity);
    dim3 grids_prepare_high(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids_prepare_high,blocks_prepare_high>>>(phi);
    hipDeviceSynchronize();
    #ifdef AMSTENCIL
        dim3 blocks_prepare_half2(uxd2,unity);
        dim3 grids_prepare_half2(1,1,unitdimX*unitdimY);
        dataprepare_half2<<<grids_prepare_half2,blocks_prepare_half2>>>(hphi);
        hipDeviceSynchronize();
        #define TOLOW __float2half2_rn
        #ifdef Monitor2
            
            for(int i=0;i<dimX*dimY;i++){
                phi_old[i]=phi[i]==0?1.0:0.0;
            }
        #endif
        dim3 blocks_detect(unitdimX>32?32:unitdimX,unitdimY>32?32:unitdimY);
        dim3 grids_detect(unitdimX>32?unitdimX/32:1,unitdimY>32?unitdimY/32:1);
        dim3 datasychronduring_blocks(uxd2,unity);
        dim3 datasychronduring_grids(1,1,unitdimX*unitdimY);
    #else
        dim3 blocks_prepare_half(unitx,unity);
        dim3 grids_prepare_half(1,1,unitdimX*unitdimY);
        dataprepare_half<<<grids_prepare_half,blocks_prepare_half>>>(hphi);
        hipDeviceSynchronize();
        #define TOLOW __float2half
    #endif
    lowprecision hdxdy=TOLOW(dxdy);
    lowprecision hdym2=TOLOW(dy*2);
    lowprecision hdxm2=TOLOW(dx*2);
    lowprecision height=TOLOW(8.0);
    lowprecision htheta0=TOLOW(theta0);
    lowprecision haniso=TOLOW(aniso);
    lowprecision hdelta=TOLOW(delta);
    lowprecision hone=TOLOW(1.0);
    lowprecision hepsilonb=TOLOW(epsilonb);
    lowprecision halpha=TOLOW(alpha);
    lowprecision hpi=TOLOW(pi);
    lowprecision hgama=TOLOW(gama);
    lowprecision hkappa=TOLOW(kappa);
    lowprecision hzpf=TOLOW(0.5);
    lowprecision hteq=TOLOW(teq);
    lowprecision hdtime=TOLOW(dtime);
    lowprecision htau=TOLOW(tau);
    dim3 blocks_mix(uxd2,unity);
    dim3 grids_mix(2,1,unitdimX*unitdimY);
    // for(int i=0;i<unitNums;i++)type_new[i]=1;
    hipEvent_t startmix,stopmix;float elapsedmix;
    for(int i=0;i<timesteps;i++){
        if(i==5){
            CHECK_ERROR(hipEventCreate(&startmix));
            CHECK_ERROR(hipEventCreate(&stopmix));
            CHECK_ERROR(hipEventRecord(startmix,0));
        }
        #ifdef AMSTENCIL
            if(i%10==0){
                #ifdef Monitor1
                get_max_diff1<<<grids_detect,blocks_detect>>>(phi,max_diff);
                #endif
                #ifdef Monitor2
                get_max_diff2<<<grids_detect,blocks_detect>>>(phi_old,phi,max_diff);
                #endif
                get_type<<<grids_detect,blocks_detect>>>(max_diff,type_old,type_new);
                hipDeviceSynchronize();
                // writetocsv("max"+to_string(i)+".csv",max_diff,unitdimX,unitdimY);
                // writetocsv("type"+to_string(i)+".csv",type_curr,unitdimX,unitdimY);
                data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(phi,tempr,hphi,htempr,type_old,type_new);
                hipDeviceSynchronize();
            }
        #endif
        kernel1_mix<<<grids_mix,blocks_mix>>>(phi,phi_old,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,htempr,htempr_lap,hphidx,hphidy,hepsilon,hepsilon_deri,hdxdy,htheta0,haniso,hone,hdxm2,hdym2,hdelta,hepsilonb,height,type_new,i);
        mix_kernel2<<<grids_mix,blocks_mix>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,hepsilon,hepsilon_deri,hphidx,hphidy,htempr,htempr_lap,hdym2,hdxm2,hgama,hteq,halpha,hpi,hone,hzpf,hdtime,htau,hkappa,type_new,i);
        hipDeviceSynchronize();
    }
    if(timesteps>5){
        CHECK_ERROR(hipEventRecord(stopmix,0));
        CHECK_ERROR(hipEventSynchronize(stopmix));
        CHECK_ERROR(hipEventElapsedTime(&elapsedmix,startmix,stopmix));
        CHECK_ERROR(hipEventDestroy(startmix));
        CHECK_ERROR(hipEventDestroy(stopmix));
    }
    ofstream ftime("time_tmp.csv");
    ftime<<elapsedmix;
    ftime.close();
    //每个程序跑五次测试性能，最后一次记录计算结果
    if(string(argv[1])=="4"){
        string path=string(argv[2]);
        data_sychro_aftercomputation<<<grids_mix,blocks_mix>>>(phi,hphi,type_new);
        hipDeviceSynchronize();
        writetocsv(path,phi,dimX,dimY);
    }

    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    CHECK_ERROR(hipFree(hphi));
    CHECK_ERROR(hipFree(hphi_lap));
    CHECK_ERROR(hipFree(htempr));
    CHECK_ERROR(hipFree(htempr_lap));
    CHECK_ERROR(hipFree(hphidx));
    CHECK_ERROR(hipFree(hphidy));
    CHECK_ERROR(hipFree(hepsilon));
    CHECK_ERROR(hipFree(hepsilon_deri));
    CHECK_ERROR(hipFree(type_new));
    CHECK_ERROR(hipFree(phi_old));
    #ifdef AMSTENCIL
        CHECK_ERROR(hipFree(type_old));
        CHECK_ERROR(hipFree(max_diff));
    #endif
    return 0;
}