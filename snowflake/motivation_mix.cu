#include "hip/hip_runtime.h"
#include "function.h"
#include "tools.h"
#include <queue>
int main(int argc ,char* argv[]){
    highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
    lowprecision *hphi,*hphi_lap,*htempr,*htempr_lap,*hphidx,*hphidy,*hepsilon,*hepsilon_deri;
    int *type_new;
    CHECK_ERROR(hipMallocManaged((void**)&hphi,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphi_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&htempr_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidx,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hphidy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hepsilon_deri,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_new,sizeof(int)*unitdimX*unitdimY));
    highprecision *phi_old;
    CHECK_ERROR(hipMallocManaged((void**)&phi_old,sizeof(highprecision)*dimX*dimY));
    #ifdef AMSTENCIL
        highprecision *max_diff;
        CHECK_ERROR(hipMallocManaged((void**)&max_diff,sizeof(highprecision)*unitdimX*unitdimY));
        #define TOLOW __float2half2_rn
        dim3 blocks_detect(unitdimX>32?32:unitdimX,unitdimY>32?32:unitdimY);
        dim3 grids_detect(unitdimX>32?unitdimX/32:1,unitdimY>32?unitdimY/32:1);
    #else
        #define TOLOW __float2half
    #endif
    dim3 blocks_prepare_high(unitx,unity);
    dim3 grids_prepare_high(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids_prepare_high,blocks_prepare_high>>>(phi);
    hipDeviceSynchronize();
    lowprecision hdxdy=TOLOW(dxdy);
    lowprecision hdym2=TOLOW(dy*2);
    lowprecision hdxm2=TOLOW(dx*2);
    lowprecision height=TOLOW(8.0);
    lowprecision htheta0=TOLOW(theta0);
    lowprecision haniso=TOLOW(aniso);
    lowprecision hdelta=TOLOW(delta);
    lowprecision hone=TOLOW(1.0);
    lowprecision hepsilonb=TOLOW(epsilonb);
    lowprecision halpha=TOLOW(alpha);
    lowprecision hpi=TOLOW(pi);
    lowprecision hgama=TOLOW(gama);
    lowprecision hkappa=TOLOW(kappa);
    lowprecision hzpf=TOLOW(0.5);
    lowprecision hteq=TOLOW(teq);
    lowprecision hdtime=TOLOW(dtime);
    lowprecision htau=TOLOW(tau);
    dim3 blocks_pure(unitx,unity);
    dim3 grids_pure(1,1,unitdimX*unitdimY);
    for(int i=0;i<timesteps;i++){
        kernel1_pure<<<grids_pure,blocks_pure>>>(phi,phi_lap,tempr,tempr_lap,phidx,phidy,epsilon,epsilon_deri);
        hipDeviceSynchronize();
        kernel2_pure<<<grids_pure,blocks_pure>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap);
        hipDeviceSynchronize();
        #if ((defined Monitor2)&&(defined AMSTENCIL))
            if(i==timesteps-10){
                motivation_monitor2_datasychr<<<grids_pure,blocks_pure>>>(phi,phi_old);
                hipDeviceSynchronize();
            }
        #endif        
    }
    dim3 blocks_mix(uxd2,unity);
    dim3 grids_mix(2,1,unitdimX*unitdimY);
    highdata_to_low<<<grids_mix,blocks_mix>>>(phi,hphi);
    highdata_to_low<<<grids_mix,blocks_mix>>>(tempr,htempr);
    #ifdef AMSTENCIL
        int highprecisionnum=unitdimX*unitdimY-unitdimX*unitdimY*ratio,sum=0;
        #ifdef Monitor1
            get_max_diff1<<<grids_detect,blocks_detect>>>(phi,max_diff);
        #endif
        #ifdef Monitor2
            get_max_diff2<<<grids_detect,blocks_detect>>>(phi_old,phi,max_diff);
        #endif
        hipDeviceSynchronize();
        int *index;
        index=new int[unitdimX*unitdimY];
        BubbleSort(max_diff,unitdimX*unitdimY,index);
        for(int j=0;j<unitdimX*unitdimY;j++)type_new[j]=1;
        queue<int> queue_index;
        for(int j=0;max_diff[j]!=0;j++){
            type_new[index[j]]=2;
            sum++;
            queue_index.push(index[j]);
            if(sum>=highprecisionnum)break;
        }
        while(!queue_index.empty()){
            int center_index=queue_index.front();
            queue_index.pop();
            for(int direct=1;direct<=8;direct++){
                if(type_new[get_neibour(center_index,direct,1)]!=2){
                    sum++;
                    type_new[get_neibour(center_index,direct,1)]=2;
                    queue_index.push(get_neibour(center_index,direct,1));
                    if(sum>=highprecisionnum)break;
                }
            }
            if(sum>=highprecisionnum)break;
        }
        delete []index;
    #endif
    for(int i=timesteps;i<timesteps+50;i++){
        kernel1_mix<<<grids_mix,blocks_mix>>>(phi,phi_old,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,htempr,htempr_lap,hphidx,hphidy,hepsilon,hepsilon_deri,hdxdy,htheta0,haniso,hone,hdxm2,hdym2,hdelta,hepsilonb,height,type_new,i);
        mix_kernel2<<<grids_mix,blocks_mix>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap,hphi,hphi_lap,hepsilon,hepsilon_deri,hphidx,hphidy,htempr,htempr_lap,hdym2,hdxm2,hgama,hteq,halpha,hpi,hone,hzpf,hdtime,htau,hkappa,type_new,i);
        hipDeviceSynchronize();

    }
    data_sychro_aftercomputation<<<grids_mix,blocks_mix>>>(phi,hphi,type_new);
    hipDeviceSynchronize();
    string paths=string(argv[1]);
    writetocsv(paths,phi,dimX,dimY);
    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    CHECK_ERROR(hipFree(hphi));
    CHECK_ERROR(hipFree(hphi_lap));
    CHECK_ERROR(hipFree(htempr));
    CHECK_ERROR(hipFree(htempr_lap));
    CHECK_ERROR(hipFree(hphidx));
    CHECK_ERROR(hipFree(hphidy));
    CHECK_ERROR(hipFree(hepsilon));
    CHECK_ERROR(hipFree(hepsilon_deri));
    CHECK_ERROR(hipFree(type_new));
    CHECK_ERROR(hipFree(phi_old));
    #ifdef AMSTENCIL
        CHECK_ERROR(hipFree(max_diff));
    #endif
    return 0;    
}