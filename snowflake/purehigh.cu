#include "function.h"
#include "tools.h"
int main(int argc,char* argv[]){
    highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
    dim3 blocks(unitx,unity);
    dim3 grids(1,1,unitdimX*unitdimY);
    dataprepare_high<<<grids,blocks>>>(phi);
    hipDeviceSynchronize();
    #ifdef Motivation
         timesteps=timesteps+50;
    #endif
    #ifdef End2end
        hipEvent_t start,stop;float elapsed;
    #endif
    for(int i=0;i<timesteps;i++){
        #ifdef End2end
            if(i==5){
                CHECK_ERROR(hipEventCreate(&start));
                CHECK_ERROR(hipEventCreate(&stop));
                CHECK_ERROR(hipEventRecord(start,0));
            }
        #endif
        kernel1_pure<<<grids,blocks>>>(phi,phi_lap,tempr,tempr_lap,phidx,phidy,epsilon,epsilon_deri);
        kernel2_pure<<<grids,blocks>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap);
        hipDeviceSynchronize();
    }
    #ifdef End2end
        if(timesteps>5){
            CHECK_ERROR(hipEventRecord(stop,0));
            CHECK_ERROR(hipEventSynchronize(stop));
            CHECK_ERROR(hipEventElapsedTime(&elapsed,start,stop));
            CHECK_ERROR(hipEventDestroy(start));
            CHECK_ERROR(hipEventDestroy(stop));
        }
        ofstream ftime("time_tmp.csv");
        ftime<<elapsed;
        ftime.close();
    #endif
    #ifdef End2end
        if(string(argv[1])=="4"){
            string path=string(argv[2]);
            writetocsv(path,phi,dimX,dimY);
        }
    #endif
    #ifdef Motivation
        string path=string(argv[1]);
        writetocsv(path,phi,dimX,dimY);
    #endif
    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    return 0;
}