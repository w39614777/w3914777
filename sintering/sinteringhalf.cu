#include "hip/hip_runtime.h"
#include "stdio.h"
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <fstream>
#include <string>
#include <mma.h>
#include ""
#include <hip/hip_fp16.h>
#include <math.h>
#include<thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
using namespace std;
using namespace nvcuda;
const int dimX=512,dimY=512;
const float coefm=5.0,coefk=2.0,coefl=5.0,dvol=0.040,dvap=0.002,dsur=16.0,dgrb=1.6,dx=0.2,dy=0.2,dxdy=dx*dy,dtime=1.0e-4;
int it=1;
float  R1=50.0,R2=25.0;//两种粒子的半径平方大小
float  Rx1=dimX/2,Rx2=Rx1,Ry1=200.0,Ry2=275.0;//两种粒子的中心坐标
#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__)
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__)
inline void checkCudaError(hipError_t error, const char *file, const int line)
{
   if (error != hipSuccess) {
      std::cerr << "CUDA CALL FAILED:" << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}
inline void checkCudaState(const char *msg, const char *file, const int line)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      std::cerr << "---" << msg << " Error---" << std::endl;
      std::cerr << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}
__global__ void half_con1(half* hcon,half* hcon_lap,half* heta1,half* heta2,half* hdfdcon,half* hdummy,
half hfour,half hdxdy,half hsix,half hzpfive,half hone,half htwo,half hsixteen,half hcoefm){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    half(*hcond)[dimX]=(half(*)[dimX])hcon;
    half(*hcon_lapd)[dimX]=(half(*)[dimX])hcon_lap;
    half(*heta1d)[dimX]=(half(*)[dimX])heta1;
    half(*heta2d)[dimX]=(half(*)[dimX])heta2;
    half(*hdfdcond)[dimX]=(half(*)[dimX])hdfdcon;
    half(*hdummyd)[dimX]=(half(*)[dimX])hdummy;
    hcon_lapd[y][x]=(hcond[y][xs1]+hcond[y][xa1]+hcond[ys1][x]+hcond[ya1][x]-hfour*hcond[y][x])/hdxdy;
    half sum2=(heta1d[y][x]*heta1d[y][x])+(heta2d[y][x]*heta2d[y][x]);
    half sum3=(heta1d[y][x]*heta1d[y][x]*heta1d[y][x])+(heta2d[y][x]*heta2d[y][x]*heta2d[y][x]);
    hdfdcond[y][x]=hone*(htwo*hcond[y][x]+hfour*sum3-hsix*sum2)-htwo*hsixteen*(hcond[y][x]*hcond[y][x])*(hone-hcond[y][x])+htwo*hsixteen*hcond[y][x]*((hone-hcond[y][x])*(hone-hcond[y][x]));
    hdummyd[y][x]=hdfdcond[y][x]-hzpfive*hcoefm*hcon_lapd[y][x];
}
__global__ void half_con2(half* hdummy,half* hdummy_lap,half* hcon,half* heta1,half* heta2,
half hfour,half hdxdy,half hone,half htwo,half hsix,half hten,half hfifteen,half hdgrb,half hdsur,half hdtime,half hdvap,half hdvol,half highbound,half lowbound){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    half(*hdummyd)[dimX]=(half(*)[dimX])hdummy;
    half(*hdummy_lapd)[dimX]=(half(*)[dimX])hdummy_lap;
    half(*hcond)[dimX]=(half(*)[dimX])hcon;
    half(*heta1d)[dimX]=(half(*)[dimX])heta1;
    half(*heta2d)[dimX]=(half(*)[dimX])heta2;
    hdummy_lapd[y][x]=(hdummyd[y][xs1]+hdummyd[y][xa1]+hdummyd[ys1][x]+hdummyd[ya1][x]-hfour*hdummyd[y][x])/hdxdy;
    half hphi=(hcond[y][x]*hcond[y][x]*hcond[y][x])*(hten-hfifteen*hcond[y][x]+hsix*(hcond[y][x]*hcond[y][x])); //插值函数
    half hsum=heta1d[y][x]*heta2d[y][x]*htwo;
    half hmobil=hdvol*hphi+hdvap*(hone-hphi)+hdsur*hcond[y][x]*(hone-hcond[y][x])+hdgrb*hsum;
    hcond[y][x]=hcond[y][x]+hdtime*hmobil*hdummy_lapd[y][x];
    hcond[y][x]=__hmin(highbound,hcond[y][x]);
    hcond[y][x]=__hmax(lowbound,hcond[y][x]);
}
__global__ void half_phi1(half* heta1,half* heta1_out,half* heta2,half* heta1_lap,half* hdfdeta1,half* hcon,
half hfour,half hdxdy,half hzpfive,half hone,half htwo,half htwelve,half hdtime,half hcoefl,half hcoefk,half highbound,half lowbound){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    half(*heta1d)[dimX]=(half(*)[dimX])heta1;
    half(*heta2d)[dimX]=(half(*)[dimX])heta2;
    half(*heta1_outd)[dimX]=(half(*)[dimX])heta1_out;
    half(*heta1_lapd)[dimX]=(half(*)[dimX])heta1_lap;
    half(*hdfdeta1d)[dimX]=(half(*)[dimX])hdfdeta1;
    half(*hcond)[dimX]=(half(*)[dimX])hcon;
    heta1_lapd[y][x]=(heta1d[y][xs1]+heta1d[y][xa1]+heta1d[ys1][x]+heta1d[ya1][x]-hfour*heta1d[y][x])/hdxdy;
    //自由能对相求导
    half hsum2=(heta1d[y][x]*heta1d[y][x])+(heta2d[y][x]*heta2d[y][x]);
    hdfdeta1d[y][x]=hone*(-htwelve*(heta1d[y][x]*heta1d[y][x])*(htwelve-hcond[y][x])+htwelve*heta1d[y][x]*(hone-hcond[y][x])+htwelve*heta1d[y][x]*hsum2);
    heta1_outd[y][x]=heta1d[y][x]-hdtime*hcoefl*(hdfdeta1d[y][x]-hzpfive*hcoefk*heta1_lapd[y][x]);
    heta1_outd[y][x]=__hmin(highbound,heta1_outd[y][x]);
    heta1_outd[y][x]=__hmax(lowbound,heta1_outd[y][x]);
}

__global__ void half_phi2(half* heta2,half* heta2_out,half* heta1,half* heta2_lap,half* hdfdeta2,half* hcon,
half hfour,half hdxdy,half hzpfive,half hone,half htwo,half htwelve,half hdtime,half hcoefl,half hcoefk,half highbound,half lowbound){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    half(*heta1d)[dimX]=(half(*)[dimX])heta1;
    half(*heta2d)[dimX]=(half(*)[dimX])heta2;
    half(*heta2_outd)[dimX]=(half(*)[dimX])heta2_out;
    half(*heta2_lapd)[dimX]=(half(*)[dimX])heta2_lap;
    half(*hdfdeta2d)[dimX]=(half(*)[dimX])hdfdeta2;
    half(*hcond)[dimX]=(half(*)[dimX])hcon;

    heta2_lapd[y][x]=(heta2d[y][xs1]+heta2d[y][xa1]+heta2d[ys1][x]+heta2d[ya1][x]-hfour*heta2d[y][x])/hdxdy;
    //自由能对相求导
    half hsum2=(heta1d[y][x]*heta1d[y][x])+(heta2d[y][x]*heta2d[y][x]);
    hdfdeta2d[y][x]=hone*(-htwelve*(heta2d[y][x]*heta2d[y][x])*(htwo-hcond[y][x])+htwelve*heta2d[y][x]*(hone-hcond[y][x])+htwelve*heta2d[y][x]*hsum2);
    heta2_outd[y][x]=heta2d[y][x]-hdtime*hcoefl*(hdfdeta2d[y][x]-hzpfive*hcoefk*heta2_lapd[y][x]);
    heta2_outd[y][x]=__hmin(highbound,heta2_outd[y][x]);
    heta2_outd[y][x]=__hmax(lowbound,heta2_outd[y][x]);
}

int main(void){
    half *hcon,*heta1,*heta2,*heta1_lap,*heta2_lap,*hcon_lap,*hdummy,*hdummy_lap,*hdfdcon,*hdfdeta1,*hdfdeta2,*heta1_out,*heta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&hcon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdummy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hcon_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdummy_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdcon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta2,sizeof(half)*dimX*dimY));
    half hcoefm=__float2half(coefm);
    half hcoefk=__float2half(coefk);
    half hcoefl=__float2half(coefl);
    half hdvol=__float2half(dvol);
    half hdvap=__float2half(dvap);
    half hdsur=__float2half(dsur);
    half hdgrb=__float2half(dgrb);
    half hdx=__float2half(dx);
    half hdy=__float2half(dy);
    half hdxdy=__float2half(dxdy);
    half hdtime=__float2half(dtime);
    half hzpfive=__float2half(0.5);
    half hone=__float2half(1.0);
    half htwo=__float2half(2.0);
    half hfour=__float2half(4.0);
    half hsix=__float2half(6.0);
    half hten=__float2half(10.0);
    half htwelve=__float2half(12.0);
    half hfifteen=__float2half(15.0);
    half hsixteen=__float2half(16.0);
    half highbound=__float2half(1);
    half lowbound=__float2half(0.0);
    for(int y=1;y<=dimY;y++){
        for(int x=1;x<=dimX;x++){
            float dis1=sqrt(pow(x-Rx1,2)+pow(y-Ry1,2));
            float dis2=sqrt(pow(x-Rx1,2)+pow(y-Ry2,2));
            if(dis1<=R1){
                hcon[(y-1)*dimX+x-1]=(half)1;
                heta1[(y-1)*dimX+x-1]=(half)1;
            }
            if(dis2<=R2){
                hcon[(y-1)*dimX+x-1]=(half)1;
                heta2[(y-1)*dimX+x-1]=(half)1;
                heta1[(y-1)*dimX+x-1]=(half)0.0;
            }
        }
    }
    dim3 blocks(32,32);
    dim3 grids(dimX/32,dimY/32);
    half_con1<<<grids,blocks>>>(hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hfour,hdxdy,hsix,hzpfive,hone,htwo,hsixteen,hcoefm);
    hipDeviceSynchronize();
    half_con2<<<grids,blocks>>>(hdummy,hdummy_lap,hcon,heta1,heta2,hfour,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound);
    hipDeviceSynchronize();
    half_phi1<<<grids,blocks>>>(heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hfour,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound);
    hipDeviceSynchronize();
    half_phi2<<<grids,blocks>>>(heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hfour,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound);
    hipDeviceSynchronize();
    swap(heta1,heta1_out);
    swap(heta2,heta2_out);
    CHECK_ERROR(hipFree(hcon));CHECK_ERROR(hipFree(hcon_lap));
    CHECK_ERROR(hipFree(heta1));CHECK_ERROR(hipFree(heta1_lap));
    CHECK_ERROR(hipFree(heta2));CHECK_ERROR(hipFree(heta2_lap));
    CHECK_ERROR(hipFree(heta1_out));CHECK_ERROR(hipFree(heta2_out));
    CHECK_ERROR(hipFree(hdummy));CHECK_ERROR(hipFree(hdummy_lap));
    CHECK_ERROR(hipFree(hdfdcon));CHECK_ERROR(hipFree(hdfdeta1));CHECK_ERROR(hipFree(hdfdeta2));
    return 0;
}