#include "hip/hip_runtime.h"
#include "tools.h"
#include <sys/stat.h> 　
#include <sys/types.h>
#include<queue>
int main(int argc,char* argv[]){
    highprecision *con,*eta1,*eta2,*eta1_lap,*eta2_lap,*con_lap,*dummy,*dummy_lap,*dfdcon,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&con,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dummy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&con_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dummy_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdcon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));
    #if((defined AMSTENCIL)||(defined GRAM1)||(defined GRAM2))
    lowprecision *hcon,*heta1,*heta2,*heta1_lap,*heta2_lap,*hcon_lap,*hdummy,*hdummy_lap,*hdfdcon,*hdfdeta1,*hdfdeta2,*heta1_out,*heta2_out;
    int *type_con;
    CHECK_ERROR(hipMallocManaged((void**)&hcon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_out,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdummy,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hcon_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta1_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&heta2_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdummy_lap,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdcon,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta1,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&hdfdeta2,sizeof(half)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&type_con,sizeof(int)*unitdimX*unitdimY));
    #endif
    #ifdef AMSTENCIL
    int *type_old;
    highprecision *max_diff_con;
    CHECK_ERROR(hipMallocManaged((void**)&type_old,sizeof(int)*unitdimX*unitdimY));
    CHECK_ERROR(hipMallocManaged((void**)&max_diff_con,sizeof(highprecision)*unitdimX*unitdimY));
    #endif

    for(int y=1;y<=dimY;y++){
        for(int x=1;x<=dimX;x++){
            float dis1=sqrt(pow(x-Rx1,2)+pow(y-Ry1,2));
            float dis2=sqrt(pow(x-Rx1,2)+pow(y-Ry2,2));
            if(dis1<=R1){
                con[(y-1)*dimX+x-1]=1;
                eta1[(y-1)*dimX+x-1]=1;
            }
            if(dis2<=R2){
                con[(y-1)*dimX+x-1]=1;
                eta2[(y-1)*dimX+x-1]=1;
                eta1[(y-1)*dimX+x-1]=0.0;
            }
        }
    }
    #if ((defined AMSTENCIL)||(defined GRAM1||(defined GRAM2)))
    highprecision *con_old;
    CHECK_ERROR(hipMallocManaged((void**)&con_old,sizeof(highprecision)*dimX*dimY));
    for(int i=0;i<dimX*dimY;i++){
        con_old[i]=con[i]==0?1.0:0.0;
    }
    #endif
    #if (defined AMSTENCIL)&&(defined Motivation)&&(defined Monitor2)
    highprecision *con_last;
    CHECK_ERROR(hipMallocManaged((void**)&con_last,sizeof(highprecision)*dimX*dimY));
    #endif 
    #if ((defined AMSTENCIL)||(defined GRAM1||(defined GRAM2)))
    dim3 blocks_dataprepare(32,32);
    dim3 grids_dataprepare(dimX/32/2,dimY/32);
    dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon);
    dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(eta1,heta1);
    dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(eta2,heta2);
    hipDeviceSynchronize();
    lowprecision hcoefm=lowprecision2highprecision(coefm);
    lowprecision hcoefk=lowprecision2highprecision(coefk);
    lowprecision hcoefl=lowprecision2highprecision(coefl);
    lowprecision hdvol=lowprecision2highprecision(dvol);
    lowprecision hdvap=lowprecision2highprecision(dvap);
    lowprecision hdsur=lowprecision2highprecision(dsur);
    lowprecision hdgrb=lowprecision2highprecision(dgrb);
    lowprecision hdx=lowprecision2highprecision(dx);
    lowprecision hdy=lowprecision2highprecision(dy);
    lowprecision hdxdy=lowprecision2highprecision(dxdy);
    lowprecision hdtime=lowprecision2highprecision(dtime);
    lowprecision hcf=lowprecision2highprecision(4.0);
    lowprecision hzpfive=lowprecision2highprecision(0.5);
    lowprecision hone=lowprecision2highprecision(1.0);
    lowprecision htwo=lowprecision2highprecision(2.0);
    lowprecision hfour=lowprecision2highprecision(4.0);
    lowprecision hsix=lowprecision2highprecision(6.0);
    lowprecision hten=lowprecision2highprecision(10.0);
    lowprecision htwelve=lowprecision2highprecision(12.0);
    lowprecision hfifteen=lowprecision2highprecision(15.0);
    lowprecision hthirtytwo=lowprecision2highprecision(32.0);
    lowprecision highbound=lowprecision2highprecision(1);
    lowprecision lowbound=lowprecision2highprecision(0.0);
    #endif

    // 线程数量设置
    #if  ((defined PURE)||(defined Motivation))
    dim3 blocks_pure(unitx,unity);
    dim3 grids_pure(1,1,unitdimX*unitdimY);
    #endif
    #if ((defined AMSTENCIL)||(defined GRAM1||(defined GRAM2))) 
    dim3 blocks(uxd2,unity);
    dim3 grids(2,1,unitdimX*unitdimY);
    #endif
    #ifdef AMSTENCIL
    dim3 blocks_detect(unitdimX>32?32:unitdimX,unitdimY>32?32:unitdimY);
    dim3 grids_detect(unitdimX>32?unitdimX/32:1,unitdimY>32?unitdimY/32:1);
    dim3 datasychronduring_blocks(uxd2,unity);
    dim3 datasychronduring_grids(1,1,unitdimX*unitdimY);
    #endif
    // 计时
    #ifdef GET_TIME
    hipEvent_t startmix,stopmix;float elapsedmix;
    #endif
    #ifdef End2end
    #ifdef GET_MonitorTIME
    float monitor_time=0;
    #endif
    for(int i=0;i<timesteps;i++){
        
        #ifdef GET_TIME
        if(i==5){
            CHECK_ERROR(hipEventCreate(&startmix));
            CHECK_ERROR(hipEventCreate(&stopmix));
            CHECK_ERROR(hipEventRecord(startmix,0));
            CHECK_ERROR(hipEventSynchronize(startmix));
        }
        #endif
        #ifdef PURE
        con1_pure<<<grids_pure,blocks_pure>>>(con,con_lap,eta1,eta2,dfdcon,dummy,i);
        hipDeviceSynchronize();
        con2_pure<<<grids_pure,blocks_pure>>>(dummy,dummy_lap,con,eta1,eta2,i);
        hipDeviceSynchronize();
        phi1_pure<<<grids_pure,blocks_pure>>>(eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,i);
        hipDeviceSynchronize();
        phi2_pure<<<grids_pure,blocks_pure>>>(eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        #endif
        #if((defined GRAM1)||(defined GRAM2))
        con1_mix<<<grids,blocks>>>(type_con,con,con_old,con_lap,eta1,eta2,dfdcon,dummy,hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hcf,hdxdy,hsix,hzpfive,hone,htwo,hfour,hthirtytwo,hcoefm,i);
        hipDeviceSynchronize();
        con2_mix<<<grids,blocks>>>(type_con,dummy,dummy_lap,con,eta1,eta2,hdummy,hdummy_lap,hcon,heta1,heta2,hcf,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi1_mix<<<grids,blocks>>>(type_con,eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi2_mix<<<grids,blocks>>>(type_con,eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
        #endif
        #ifdef AMSTENCIL
        if(i%10==0){
            #ifdef GET_MonitorTIME
            hipEvent_t startmonitor,stopmonitor;float elapsedmonitor=0;
            if(i>=5){
                CHECK_ERROR(hipEventCreate(&startmonitor));
                CHECK_ERROR(hipEventCreate(&stopmonitor));
                CHECK_ERROR(hipEventRecord(startmonitor,0));
                CHECK_ERROR(hipEventSynchronize(startmonitor));
            }
            #endif
            #ifdef Monitor1
            get_max_diff1<<<grids_detect,blocks_detect>>>(con,max_diff_con);
            #endif
            #ifdef Monitor2
            get_max_diff2<<<grids_detect,blocks_detect>>>(con_old,con,max_diff_con);
            #endif
            get_type<<<grids_detect,blocks_detect>>>(max_diff_con,type_old,type_con);
            hipDeviceSynchronize();
            data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(con,eta1,eta2,hcon,heta1,heta2,type_old,type_con);
            hipDeviceSynchronize();
            #ifdef GET_MonitorTIME
            if(i>=5){
                CHECK_ERROR(hipEventRecord(stopmonitor,0));
                CHECK_ERROR(hipEventSynchronize(stopmonitor));
                CHECK_ERROR(hipEventElapsedTime(&elapsedmonitor,startmonitor,stopmonitor));
                CHECK_ERROR(hipEventDestroy(startmonitor));
                CHECK_ERROR(hipEventDestroy(stopmonitor));
            }
            monitor_time=monitor_time+elapsedmonitor;
            #endif
        }
        con1_mix<<<grids,blocks>>>(type_con,con,con_old,con_lap,eta1,eta2,dfdcon,dummy,hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hcf,hdxdy,hsix,hzpfive,hone,htwo,hfour,hthirtytwo,hcoefm,i);
        hipDeviceSynchronize();
        con2_mix<<<grids,blocks>>>(type_con,dummy,dummy_lap,con,eta1,eta2,hdummy,hdummy_lap,hcon,heta1,heta2,hcf,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi1_mix<<<grids,blocks>>>(type_con,eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi2_mix<<<grids,blocks>>>(type_con,eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
        #endif
    }
    #ifdef GET_MonitorTIME
    ofstream ftime("time_tmp.csv");
    ftime<<monitor_time;
    ftime.close();
    #endif
    #endif
    #ifdef GET_TIME
    if(timesteps>5){
        CHECK_ERROR(hipEventRecord(stopmix,0));
        CHECK_ERROR(hipEventSynchronize(stopmix));
        CHECK_ERROR(hipEventElapsedTime(&elapsedmix,startmix,stopmix));
        CHECK_ERROR(hipEventDestroy(startmix));
        CHECK_ERROR(hipEventDestroy(stopmix));
    }
    ofstream ftime("time_tmp.csv");
    ftime<<elapsedmix;
    ftime.close();
    #endif
    #ifdef Motivation
    for(int i=0;i<timesteps;i++){
        con1_pure<<<grids_pure,blocks_pure>>>(con,con_lap,eta1,eta2,dfdcon,dummy,i);
        hipDeviceSynchronize();
        con2_pure<<<grids_pure,blocks_pure>>>(dummy,dummy_lap,con,eta1,eta2,i);
        hipDeviceSynchronize();
        phi1_pure<<<grids_pure,blocks_pure>>>(eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,i);
        hipDeviceSynchronize();
        phi2_pure<<<grids_pure,blocks_pure>>>(eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out); 
        #if ((defined Monitor2)&&(defined AMSTENCIL))
        if(i==timesteps-10){
            for(int j=0;j<dimX*dimY;j++)con_last[j]=con[j];
        }
        #endif
    }
    for(int i=timesteps;i<timesteps+50;i++){
        #ifdef PURE
        con1_pure<<<grids_pure,blocks_pure>>>(con,con_lap,eta1,eta2,dfdcon,dummy,i);
        hipDeviceSynchronize();
        con2_pure<<<grids_pure,blocks_pure>>>(dummy,dummy_lap,con,eta1,eta2,i);
        hipDeviceSynchronize();
        phi1_pure<<<grids_pure,blocks_pure>>>(eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,i);
        hipDeviceSynchronize();
        phi2_pure<<<grids_pure,blocks_pure>>>(eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out); 
        #endif
        #if((defined GRAM1)||(defined GRAM2))
        if(i==timesteps){
            dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon);
            dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(eta1,heta1);
            dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(eta2,heta2);
        }
        con1_mix<<<grids,blocks>>>(type_con,con,con_old,con_lap,eta1,eta2,dfdcon,dummy,hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hcf,hdxdy,hsix,hzpfive,hone,htwo,hfour,hthirtytwo,hcoefm,i);
        hipDeviceSynchronize();
        con2_mix<<<grids,blocks>>>(type_con,dummy,dummy_lap,con,eta1,eta2,hdummy,hdummy_lap,hcon,heta1,heta2,hcf,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi1_mix<<<grids,blocks>>>(type_con,eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi2_mix<<<grids,blocks>>>(type_con,eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
        #endif
        #ifdef AMSTENCIL
        if(i==timesteps){
            int *index;
            index=new int[unitdimX*unitdimY];
            dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon);
            dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(eta1,heta1);
            dataprepare<<<grids_dataprepare,blocks_dataprepare>>>(eta2,heta2);
            int highprecisionnum=unitdimX*unitdimY-unitdimX*unitdimY*ratio,sum=0;
            #ifdef Monitor1
            get_max_diff1<<<grids_detect,blocks_detect>>>(con,max_diff_con);
            #endif
            #ifdef Monitor2
            get_max_diff2<<<grids_detect,blocks_detect>>>(con_last,con,max_diff_con);
            #endif
            hipDeviceSynchronize();
            BubbleSort(max_diff_con,unitdimX*unitdimY,index);
            for(int j=0;j<unitdimX*unitdimY;j++)type_con[j]=1;
            queue<int> queue_index;
            for(int j=0;max_diff_con[j]!=0;j++){
                type_con[index[j]]=2;
                sum++;
                queue_index.push(index[j]);
                if(sum>=highprecisionnum)break;
            }
            while(!queue_index.empty()){
                int center_index=queue_index.front();
                queue_index.pop();
                for(int direct=1;direct<=8;direct++){
                    if(type_con[get_neibour(center_index,direct,1)]!=2){
                        sum++;
                        type_con[get_neibour(center_index,direct,1)]=2;
                        queue_index.push(get_neibour(center_index,direct,1));
                        if(sum>=highprecisionnum)break;
                    }
                }
                if(sum>=highprecisionnum)break;
            }
            delete []index;
        }

        con1_mix<<<grids,blocks>>>(type_con,con,con_old,con_lap,eta1,eta2,dfdcon,dummy,hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hcf,hdxdy,hsix,hzpfive,hone,htwo,hfour,hthirtytwo,hcoefm,i);
        hipDeviceSynchronize();
        con2_mix<<<grids,blocks>>>(type_con,dummy,dummy_lap,con,eta1,eta2,hdummy,hdummy_lap,hcon,heta1,heta2,hcf,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi1_mix<<<grids,blocks>>>(type_con,eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi2_mix<<<grids,blocks>>>(type_con,eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
        #endif
    }
    #endif

    #ifdef monitor_conversion_independent
    #ifdef Monitor1
    get_max_diff1<<<grids_detect,blocks_detect>>>(con,max_diff_con);
    #endif
    #ifdef Monitor2
    get_max_diff2<<<grids_detect,blocks_detect>>>(con_old,con,max_diff_con);
    #endif
    get_type<<<grids_detect,blocks_detect>>>(max_diff_con,type_old,type_con);
    hipDeviceSynchronize();
    data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(con,eta1,eta2,hcon,heta1,heta2,type_old,type_con);
    hipDeviceSynchronize();
    for(int i=0;i<5;i++){
        #ifdef Monitor2
        monitor2_lastdata_store<<<grids,blocks>>>(con,con_old,i);
        hipDeviceSynchronize();
        #endif

        con1_mix<<<grids,blocks>>>(type_con,con,con_old,con_lap,eta1,eta2,dfdcon,dummy,hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hcf,hdxdy,hsix,hzpfive,hone,htwo,hfour,hthirtytwo,hcoefm,i);
        hipDeviceSynchronize();
        con1_conversion<<<grids,blocks>>>(type_con,dummy,hdummy);
        hipDeviceSynchronize();

        con2_mix<<<grids,blocks>>>(type_con,dummy,dummy_lap,con,eta1,eta2,hdummy,hdummy_lap,hcon,heta1,heta2,hcf,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound,i);
        hipDeviceSynchronize();
        con2_conversion<<<grids,blocks>>>(type_con,con,hcon,i);
        hipDeviceSynchronize();

        phi1_mix<<<grids,blocks>>>(type_con,eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        phi1_conversion<<<grids,blocks>>>(type_con,eta1_out,heta1_out,i);
        hipDeviceSynchronize();

        phi2_mix<<<grids,blocks>>>(type_con,eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        phi2_conversion<<<grids,blocks>>>(type_con,eta2_out,heta2_out,i);
        hipDeviceSynchronize();

        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
    }
    hipEvent_t kernel_start,kernel_end;float elapsed_kernel=0;
    CHECK_ERROR(hipEventCreate(&kernel_start));
    CHECK_ERROR(hipEventCreate(&kernel_end));
    float total_time=0;
    for(int i=5;i<timesteps;i++){
        if(i%10==0){
            #ifdef Monitor1
            get_max_diff1<<<grids_detect,blocks_detect>>>(con,max_diff_con);
            #endif
            #ifdef Monitor2
            get_max_diff2<<<grids_detect,blocks_detect>>>(con_old,con,max_diff_con);
            #endif
            get_type<<<grids_detect,blocks_detect>>>(max_diff_con,type_old,type_con);
            hipDeviceSynchronize();
            data_sychro_duringcomputation<<<datasychronduring_grids,datasychronduring_blocks>>>(con,eta1,eta2,hcon,heta1,heta2,type_old,type_con);
            hipDeviceSynchronize();
        }
        #ifdef Monitor2
        monitor2_lastdata_store<<<grids,blocks>>>(con,con_old,i);
        hipDeviceSynchronize();
        #endif

        CHECK_ERROR(hipEventRecord(kernel_start,0));
        con1_mix<<<grids,blocks>>>(type_con,con,con_old,con_lap,eta1,eta2,dfdcon,dummy,hcon,hcon_lap,heta1,heta2,hdfdcon,hdummy,hcf,hdxdy,hsix,hzpfive,hone,htwo,hfour,hthirtytwo,hcoefm,i);
        hipDeviceSynchronize();
        CHECK_ERROR(hipEventRecord(kernel_end,0));
        CHECK_ERROR(hipEventSynchronize(kernel_end));
        CHECK_ERROR(hipEventElapsedTime(&elapsed_kernel,kernel_start,kernel_end));
        total_time+=elapsed_kernel;
        con1_conversion<<<grids,blocks>>>(type_con,dummy,hdummy);
        hipDeviceSynchronize();

        CHECK_ERROR(hipEventRecord(kernel_start,0));
        con2_mix<<<grids,blocks>>>(type_con,dummy,dummy_lap,con,eta1,eta2,hdummy,hdummy_lap,hcon,heta1,heta2,hcf,hdxdy,hone,htwo,hsix,hten,hfifteen,hdgrb,hdsur,hdtime,hdvap,hdvol,highbound,lowbound,i);
        hipDeviceSynchronize();
        CHECK_ERROR(hipEventRecord(kernel_end,0));
        CHECK_ERROR(hipEventSynchronize(kernel_end));
        CHECK_ERROR(hipEventElapsedTime(&elapsed_kernel,kernel_start,kernel_end));
        total_time+=elapsed_kernel;
        con2_conversion<<<grids,blocks>>>(type_con,con,hcon,i);
        hipDeviceSynchronize();

        CHECK_ERROR(hipEventRecord(kernel_start,0));
        phi1_mix<<<grids,blocks>>>(type_con,eta1,eta1_out,eta2,eta1_lap,dfdeta1,con,heta1,heta1_out,heta2,heta1_lap,hdfdeta1,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        hipDeviceSynchronize();
        CHECK_ERROR(hipEventRecord(kernel_end,0));
        CHECK_ERROR(hipEventSynchronize(kernel_end));
        CHECK_ERROR(hipEventElapsedTime(&elapsed_kernel,kernel_start,kernel_end));
        total_time+=elapsed_kernel;
        phi1_conversion<<<grids,blocks>>>(type_con,eta1_out,heta1_out,i);
        hipDeviceSynchronize();

        CHECK_ERROR(hipEventRecord(kernel_start,0));
        phi2_mix<<<grids,blocks>>>(type_con,eta2,eta2_out,eta1,eta2_lap,dfdeta2,con,heta2,heta2_out,heta1,heta2_lap,hdfdeta2,hcon,hcf,hdxdy,hzpfive,hone,htwo,htwelve,hdtime,hcoefl,hcoefk,highbound,lowbound,i);
        CHECK_ERROR(hipEventRecord(kernel_end,0));
        CHECK_ERROR(hipEventSynchronize(kernel_end));
        CHECK_ERROR(hipEventElapsedTime(&elapsed_kernel,kernel_start,kernel_end));
        total_time+=elapsed_kernel;
        phi2_conversion<<<grids,blocks>>>(type_con,eta2_out,heta2_out,i);
        hipDeviceSynchronize();

        swap(eta1,eta1_out);
        swap(eta2,eta2_out);
        swap(heta1,heta1_out);
        swap(heta2,heta2_out);
    }
    CHECK_ERROR(hipEventDestroy(kernel_start));
    CHECK_ERROR(hipEventDestroy(kernel_end));
    ofstream ftime("time_tmp.csv");
    ftime<<total_time;
    ftime.close();
    #endif

    #ifdef GET_RESULT
    string s="./result/";
        if(access(s.c_str(),0777)==-1){
            mkdir(s.c_str(),0777);
        }

        #ifdef End2end
            s=s+"end2end/";
            if(access(s.c_str(),0777)==-1){
                mkdir(s.c_str(),0777);
            }
            s=s+string(argv[2])+"/";
            if(access(s.c_str(),0777)==-1){
                mkdir(s.c_str(),0777);
            }
            #ifdef PURE
                s=s+"pure/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+".csv",con,dimX,dimY); 
            #endif
            #ifdef AMSTENCIL
                data_sychro_aftercomputation<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon,type_con);
                hipDeviceSynchronize();
                s=s+"amstencil/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                #ifdef Monitor1
                s=s+"monitor1/";
                string thresholdstr = to_string(threshold);
                #endif
                #ifdef Monitor2
                s=s+"monitor2/";
                ostringstream o_tmp;
                o_tmp<<threshold;
                istringstream i_tmp(o_tmp.str());
                string thresholdstr;
                i_tmp>>thresholdstr;
                #endif
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+"_"+thresholdstr+".csv",con,dimX,dimY);
            #endif
            
            #ifdef GRAM1
                data_sychro_aftercomputation<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon,type_con);
                hipDeviceSynchronize();
                s=s+"gram1/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+"_"+to_string(ratio)+".csv",con,dimX,dimY);
            #endif

            #ifdef GRAM2
                data_sychro_aftercomputation<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon,type_con);
                hipDeviceSynchronize();
                s=s+"gram2/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+"_"+to_string(ratio)+".csv",con,dimX,dimY);
            #endif
        #endif
        #ifdef Motivation
            s=s+"motivation/";
            if(access(s.c_str(),0777)==-1){
                mkdir(s.c_str(),0777);
            }
            s=s+string(argv[2])+"/";
            if(access(s.c_str(),0777)==-1){
                mkdir(s.c_str(),0777);
            }
            #ifdef PURE
                s=s+"pure/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+".csv",con,dimX,dimY); 
            #endif
            #ifdef AMSTENCIL
                data_sychro_aftercomputation<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon,type_con);
                hipDeviceSynchronize();
                s=s+"amstencil/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                #ifdef Monitor1
                s=s+"monitor1/";
                #endif
                #ifdef Monitor2
                s=s+"monitor2/";
                #endif
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+"_"+to_string(ratio)+".csv",con,dimX,dimY);
            #endif
        
            #ifdef GRAM1
                data_sychro_aftercomputation<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon,type_con);
                hipDeviceSynchronize();
                s=s+"gram1/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+"_"+to_string(ratio)+".csv",con,dimX,dimY);
            #endif

            #ifdef GRAM2
                data_sychro_aftercomputation<<<grids_dataprepare,blocks_dataprepare>>>(con,hcon,type_con);
                hipDeviceSynchronize();
                s=s+"gram2/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                s=s+"para"+string(argv[1])+"/";
                if(access(s.c_str(),0777)==-1){
                    mkdir(s.c_str(),0777);
                }
                writetocsv(s+to_string(timesteps)+"_"+to_string(ratio)+".csv",con,dimX,dimY);
            #endif
        #endif
    #endif
    CHECK_ERROR(hipFree(con));
    CHECK_ERROR(hipFree(eta1));
    CHECK_ERROR(hipFree(eta2));
    CHECK_ERROR(hipFree(eta1_out));
    CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(dummy));
    CHECK_ERROR(hipFree(con_lap));
    CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(dummy_lap));
    CHECK_ERROR(hipFree(dfdcon));
    CHECK_ERROR(hipFree(dfdeta1));
    CHECK_ERROR(hipFree(dfdeta2));
    #if((defined AMSTENCIL)||(defined GRAM1)||(defined GRAM2))
    CHECK_ERROR(hipFree(hcon));
    CHECK_ERROR(hipFree(heta1));
    CHECK_ERROR(hipFree(heta2));
    CHECK_ERROR(hipFree(heta1_out));
    CHECK_ERROR(hipFree(heta2_out));
    CHECK_ERROR(hipFree(hdummy));
    CHECK_ERROR(hipFree(hcon_lap));
    CHECK_ERROR(hipFree(heta1_lap));
    CHECK_ERROR(hipFree(heta2_lap));
    CHECK_ERROR(hipFree(hdummy_lap));
    CHECK_ERROR(hipFree(hdfdcon));
    CHECK_ERROR(hipFree(hdfdeta1));
    CHECK_ERROR(hipFree(hdfdeta2));
    #endif
    #ifdef AMSTENCIL
    CHECK_ERROR(hipFree(type_con));
    CHECK_ERROR(hipFree(type_old));
    CHECK_ERROR(hipFree(max_diff_con));
    #endif
    return 0;

}