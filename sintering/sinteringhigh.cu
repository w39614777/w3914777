#include "hip/hip_runtime.h"
#include "stdio.h"
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <fstream>
#include <string>
#include <mma.h>
#include ""
#include <hip/hip_fp16.h>
#include <math.h>
#include<thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
using namespace std;
using namespace nvcuda;
typedef double highprecision;
const int dimX=512,dimY=512;
const int unitx=16,unity=16,unitdimX=dimX/unitx,unitdimY=dimY/unity,uxd2=unitx/2,uxd2s1=uxd2-1,uxs1=unitx-1,uys1=unity-1,dimXd2=dimX/2,unitNums=unitdimX*unitdimY,lowprecison_dimX=dimX/2;
const highprecision coefm=5.0,coefk=2.0,coefl=5.0,dvol=0.040,dvap=0.002,dsur=32,dgrb=1.6,dx=0.5,dy=0.5,dxdy=dx*dy,dtime=1.0e-4;
highprecision R1=50,R2=R1/2;
highprecision Ry1=240,Ry2=Ry1+R1+R2,Rx1=dimX/2+8;
#define CHECK_ERROR(error) checkCudaError(error, __FILE__, __LINE__)
#define CHECK_STATE(msg) checkCudaState(msg, __FILE__, __LINE__)
inline void checkCudaError(hipError_t error, const char *file, const int line)
{
   if (error != hipSuccess) {
      std::cerr << "CUDA CALL FAILED:" << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}
inline void checkCudaState(const char *msg, const char *file, const int line)
{
   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      std::cerr << "---" << msg << " Error---" << std::endl;
      std::cerr << file << "( " << line << ")- " << hipGetErrorString(error) << std::endl;
      exit(EXIT_FAILURE);
   }
}
__global__ void con1(highprecision* hcon,highprecision* hcon_lap,highprecision* heta1,highprecision* heta2,highprecision* hdfdcon,highprecision* hdummy){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    highprecision(*hcond)[dimX]=(highprecision(*)[dimX])hcon;
    highprecision(*hcon_lapd)[dimX]=(highprecision(*)[dimX])hcon_lap;
    highprecision(*heta1d)[dimX]=(highprecision(*)[dimX])heta1;
    highprecision(*heta2d)[dimX]=(highprecision(*)[dimX])heta2;
    highprecision(*hdfdcond)[dimX]=(highprecision(*)[dimX])hdfdcon;
    highprecision(*hdummyd)[dimX]=(highprecision(*)[dimX])hdummy;
    hcon_lapd[y][x]=(hcond[y][xs1]+hcond[y][xa1]+hcond[ys1][x]+hcond[ya1][x]-4.0*hcond[y][x])/dxdy;
    highprecision sum2=pow(heta1d[y][x],2)+pow(heta2d[y][x],2);
    highprecision sum3=pow(heta1d[y][x],3)+pow(heta2d[y][x],3);
    hdfdcond[y][x]=1.0*(2.0*hcond[y][x]+4.0*sum3-6.0*sum2)-2.0*16.0*pow(hcond[y][x],2)*(1.0-hcond[y][x])+2.0*16.0*hcond[y][x]*pow(1.0-hcond[y][x],2);
    hdummyd[y][x]=hdfdcond[y][x]-0.5*coefm*hcon_lapd[y][x];
}
__global__ void con2(highprecision* dummy,highprecision* dummy_lap,highprecision* con,highprecision* eta1,highprecision* eta2){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    highprecision(*dummyd)[dimX]=(highprecision(*)[dimX])dummy;
    highprecision(*dummy_lapd)[dimX]=(highprecision(*)[dimX])dummy_lap;
    highprecision(*cond)[dimX]=(highprecision(*)[dimX])con;
    highprecision(*eta1d)[dimX]=(highprecision(*)[dimX])eta1;
    highprecision(*eta2d)[dimX]=(highprecision(*)[dimX])eta2;
    dummy_lapd[y][x]=(dummyd[y][xs1]+dummyd[y][xa1]+dummyd[ys1][x]+dummyd[ya1][x]-4.0*dummyd[y][x])/dxdy;
    highprecision phi=pow(cond[y][x],3)*(10.0-15.0*cond[y][x]+6.0*pow(cond[y][x],2)); //插值函数
    highprecision sum=eta1d[y][x]*eta2d[y][x]*2;
    highprecision mobil=dvol*phi+dvap*(1.0-phi)+dsur*cond[y][x]*(1.0-cond[y][x])+dgrb*sum;
    cond[y][x]=cond[y][x]+dtime*mobil*dummy_lapd[y][x];
    if(cond[y][x]>=1) cond[y][x]=1;
    else if(cond[y][x]<0)cond[y][x]=0;
}
__global__ void phi1(highprecision* eta1,highprecision* eta1_out,highprecision* eta2,highprecision* eta1_lap,highprecision* dfdeta1,highprecision* con){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    highprecision(*eta1d)[dimX]=(highprecision(*)[dimX])eta1;
    highprecision(*eta2d)[dimX]=(highprecision(*)[dimX])eta2;
    highprecision(*eta1_outd)[dimX]=(highprecision(*)[dimX])eta1_out;
    highprecision(*eta1_lapd)[dimX]=(highprecision(*)[dimX])eta1_lap;
    highprecision(*dfdeta1d)[dimX]=(highprecision(*)[dimX])dfdeta1;
    highprecision(*cond)[dimX]=(highprecision(*)[dimX])con;
    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    //lap算子
    eta1_lapd[y][x]=(eta1d[y][xs1]+eta1d[y][xa1]+eta1d[ys1][x]+eta1d[ya1][x]-4.0*eta1d[y][x])/dxdy;
    //自由能对相求导
    highprecision sum2=pow(eta1d[y][x],2)+pow(eta2d[y][x],2);
    dfdeta1d[y][x]=1.0*(-12.0*pow(eta1d[y][x],2)*(2.0-cond[y][x])+12.0*eta1d[y][x]*(1.0-cond[y][x])+12.0*eta1d[y][x]*sum2);
    eta1_outd[y][x]=eta1d[y][x]-dtime*coefl*(dfdeta1d[y][x]-0.5*coefk*eta1_lapd[y][x]);
    if(eta1_outd[y][x]>=1) eta1_outd[y][x]=1;
    else if(eta1_outd[y][x]<0)eta1_outd[y][x]=0;
}

__global__ void phi2(highprecision* eta2,highprecision* eta2_out,highprecision* eta1,highprecision* eta2_lap,highprecision* dfdeta2,highprecision* con){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    highprecision(*eta1d)[dimX]=(highprecision(*)[dimX])eta1;
    highprecision(*eta2d)[dimX]=(highprecision(*)[dimX])eta2;
    highprecision(*eta2_outd)[dimX]=(highprecision(*)[dimX])eta2_out;
    highprecision(*eta2_lapd)[dimX]=(highprecision(*)[dimX])eta2_lap;
    highprecision(*dfdeta2d)[dimX]=(highprecision(*)[dimX])dfdeta2;
    highprecision(*cond)[dimX]=(highprecision(*)[dimX])con;

    int xs1=x>0?x-1:dimX-1;int ys1=y>0?y-1:dimY-1;
    int xa1=x<dimX-1?x+1:0;int ya1=y<dimY-1?y+1:0;
    //lap算子
    eta2_lapd[y][x]=(eta2d[y][xs1]+eta2d[y][xa1]+eta2d[ys1][x]+eta2d[ya1][x]-4.0*eta2d[y][x])/dxdy;
    //自由能对相求导
    highprecision sum2=pow(eta1d[y][x],2)+pow(eta2d[y][x],2);
    dfdeta2d[y][x]=1.0*(-12.0*pow(eta2d[y][x],2)*(2.0-cond[y][x])+12.0*eta2d[y][x]*(1.0-cond[y][x])+12.0*eta2d[y][x]*sum2);
    eta2_outd[y][x]=eta2d[y][x]-dtime*coefl*(dfdeta2d[y][x]-0.5*coefk*eta2_lapd[y][x]);
    if(eta2_outd[y][x]>=1) eta2_outd[y][x]=1;
    else if(eta2_outd[y][x]<0)eta2_outd[y][x]=0;
}

int main(void){
    highprecision *con,*eta1,*eta2,*eta1_lap,*eta2_lap,*con_lap,*dummy,*dummy_lap,*dfdcon,*dfdeta1,*dfdeta2,*eta1_out,*eta2_out;
    CHECK_ERROR(hipMallocManaged((void**)&con,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_out,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dummy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&con_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta1_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&eta2_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dummy_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdcon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta1,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&dfdeta2,sizeof(highprecision)*dimX*dimY));
    for(int y=1;y<=dimY;y++){
        for(int x=1;x<=dimX;x++){
            float dis1=sqrt(pow(x-Rx1,2)+pow(y-Ry1,2));
            float dis2=sqrt(pow(x-Rx1,2)+pow(y-Ry2,2));
            if(dis1<=R1){
                con[(y-1)*dimX+x-1]=1;
                eta1[(y-1)*dimX+x-1]=1;
            }
            if(dis2<=R2){
                con[(y-1)*dimX+x-1]=1;
                eta2[(y-1)*dimX+x-1]=1;
                eta1[(y-1)*dimX+x-1]=0.0;
            }
        }
    }
    dim3 blocks(32,32);
    dim3 grids(dimX/32,dimY/32);
    con1<<<grids,blocks>>>(con,con_lap,eta1,eta2,dfdcon,dummy);
    hipDeviceSynchronize();
    con2<<<grids,blocks>>>(dummy,dummy_lap,con,eta1,eta2);
    hipDeviceSynchronize();
    phi1<<<grids,blocks>>>(eta1,eta1_out,eta2,eta1_lap,dfdeta1,con);
    hipDeviceSynchronize();
    phi2<<<grids,blocks>>>(eta2,eta2_out,eta1,eta2_lap,dfdeta2,con);
    hipDeviceSynchronize();
    swap(eta1,eta1_out);
    swap(eta2,eta2_out);
    CHECK_ERROR(hipFree(con));CHECK_ERROR(hipFree(con_lap));
    CHECK_ERROR(hipFree(eta1));CHECK_ERROR(hipFree(eta1_lap));
    CHECK_ERROR(hipFree(eta2));CHECK_ERROR(hipFree(eta2_lap));
    CHECK_ERROR(hipFree(eta1_out));CHECK_ERROR(hipFree(eta2_out));
    CHECK_ERROR(hipFree(dummy));CHECK_ERROR(hipFree(dummy_lap));
    CHECK_ERROR(hipFree(dfdcon));CHECK_ERROR(hipFree(dfdeta1));CHECK_ERROR(hipFree(dfdeta2));
    return 0;
}